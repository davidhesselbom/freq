#include "hip/hip_runtime.h"
#include "resamplecuda.cu.h"
#include "cuda_vector_types_op.h"
#include "drawnwaveformkerneldef.h"
#include <stdio.h>


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, int readstop, float scaling, float writeposoffs );


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform_with_lines(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, int readstop, float scaling, float writeposoffs );


void drawWaveform(
        DataStorage<float>::Ptr in_waveformp,
        Tfr::ChunkData::Ptr out_waveform_matrixp,
        float blob, int readstop, float maxValue, float writeposoffs )
{
    CudaGlobalReadOnly<float, 1> in_waveform = CudaGlobalStorage::ReadOnly<1>( in_waveformp );
    CudaGlobalReadWrite<float2, 2> out_waveform_matrix( CudaGlobalStorage::ReadWrite<2>( out_waveform_matrixp ).getCudaPitchedPtr() );

    int w = out_waveform_matrixp->size().width;
    dim3 block(drawWaveform_BLOCK_SIZE, 1, 1);
    dim3 grid(int_div_ceil(w, block.x), 1, 1);

    if(grid.x>65535) {
        printf("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    if (blob > 1)
    {
        kernel_draw_waveform<<<grid, block, 0, 0>>>( in_waveform, out_waveform_matrix, blob, readstop, 1.f/maxValue, writeposoffs );
    }
    else
    {
        kernel_draw_waveform_with_lines<<<grid, block, 0, 0>>>( in_waveform, out_waveform_matrix, blob, readstop, 1.f/maxValue, writeposoffs );
    }
}


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, int readstop, float scaling, float writeposoffs )
{
    int writePos_x = blockIdx.x * blockDim.x + threadIdx.x;

    draw_waveform_elem(
            writePos_x,
            in_waveform,
            out_waveform_matrix, blob, readstop, scaling, writeposoffs );
}


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform_with_lines(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, int readstop, float scaling, float writeposoffs )
{
    int writePos_x = blockIdx.x * blockDim.x + threadIdx.x;

    draw_waveform_with_lines_elem(
            writePos_x,
            in_waveform,
            out_waveform_matrix, blob, readstop, scaling, writeposoffs );
}
