#include "hip/hip_runtime.h"
#ifndef WAVELETKERNEL_CUDA_CU
#define WAVELETKERNEL_CUDA_CU

#include <stdio.h>

#include "resamplecuda.cu.h"
#include "cuda_vector_types_op.h"
#include "waveletkerneldef.h"


__global__ void kernel_compute_wavelet_coefficients( float2* in_waveform_ft, float2* out_wavelet_ft, int nFrequencyBins, int nScales, float first_j, float v, float sigma_t0, float normalization_factor );
__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, DataStorageSize numElem );
//__global__ void kernel_inverse_ellipse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, int n_valid_samples );
//__global__ void kernel_inverse_box( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, int n_valid_samples );
__global__ void kernel_clamp( cudaPitchedPtrType<float2> in_wt, size_t sample_offset, cudaPitchedPtrType<float2> out_clamped_wt );

static const char* gLastError = 0;

const char* wtGetError() {
    const char* r = gLastError;
    gLastError = 0;
    return r;
}

void setError(const char* staticErrorMessage) {
    gLastError = staticErrorMessage;
    printf("%s\n", staticErrorMessage);
}

#define TOSTR2(x) #x
#define TOSTR(x) TOSTR2(x)

#ifdef _MSC_VER
    #define setError(x) setError(__FUNCTION__ ": " x)
#endif


void wtCompute(
        DataStorage<Tfr::ChunkElement>::Ptr in_waveform_ftp,
        Tfr::ChunkData::Ptr out_wavelet_ftp,
        float fs,
        float /*minHz*/,
        float maxHz,
        int half_sizes,
        float scales_per_octave,
        float sigma_t0,
        float normalization_factor )
{
    CudaGlobalStorage::useCudaPitch( out_wavelet_ftp, false );

    Tfr::ChunkElement* in_waveform_ft = CudaGlobalStorage::ReadOnly<1>( in_waveform_ftp ).device_ptr();
    Tfr::ChunkElement* out_wavelet_ft = CudaGlobalStorage::WriteAll<2>( out_wavelet_ftp ).device_ptr();

    DataStorageSize size = out_wavelet_ftp->size();

//    nyquist = FS/2
//    a = 2 ^ (1/v)
//    aj = a^j
//    hz = fs/2/aj
//    maxHz = fs/2/(a^j)
//    (a^j) = fs/2/maxHz
//    exp(log(a)*j) = fs/2/maxHz
//    j = log(fs/2/maxHz) / log(a)
//    const float log2_a = log2f(2.f) / v = 1.f/v; // a = 2^(1/v)
    float j = (log2f(fs/2) - log2f(maxHz)) * scales_per_octave;
    float first_scale = j;

    j = floor(j+0.5f);

    if (j<0) {
        printf("j = %g, maxHz = %g, fs = %g\n", j, maxHz, fs);
        setError("Invalid argument, maxHz must be less than or equal to fs/2.");
        return;
    }

    int nFrequencyBins = size.width;
    const int N = nFrequencyBins;

    dim3 block(64,1,1);
    dim3 grid( int_div_ceil(N, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    const float pi = 3.141592654f;
    normalization_factor *= sqrt( 4*pi*sigma_t0 );
    normalization_factor *= 2.f/(float)(nFrequencyBins*half_sizes);

    kernel_compute_wavelet_coefficients<<<grid, block, 0>>>(
            (float2*)in_waveform_ft,
            (float2*)out_wavelet_ft,
            size.width, size.height,
            first_scale,
            scales_per_octave,
            sigma_t0,
            normalization_factor );
}


__global__ void kernel_compute_wavelet_coefficients(
        float2* in_waveform_ft,
        float2* out_wavelet_ft,
        int nFrequencyBins, int nScales, float first_scale, float v, float sigma_t0,
        float normalization_factor )
{
    // Which frequency bin in the discrete fourier transform this thread
    // should work with
    const int
            w_bin = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Negative frequencies are defined as 0 and are not stored in in_waveform_ft
    if (w_bin<nFrequencyBins/2)
    {
        compute_wavelet_coefficients_elem(
                w_bin,
                in_waveform_ft,
                out_wavelet_ft,
                nFrequencyBins,
                nScales,
                first_scale,
                v,
                sigma_t0,
                normalization_factor);
    }
    else if (w_bin<nFrequencyBins)
    {
        for( int j=0; j<nScales; j++)
        {
            int offset = (nScales-1-j)*nFrequencyBins;
            out_wavelet_ft[offset + w_bin] = make_float2(0,0);
        }
    }
}


void wtInverse( Tfr::ChunkData::Ptr in_waveletp, DataStorage<float>::Ptr out_inverse_waveform, DataStorageSize x )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(x.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    // kernel_inverse<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem );
    kernel_inverse<<<grid, block>>>(
            (float2*)CudaGlobalStorage::ReadOnly<2>(in_waveletp).device_ptr(),
            CudaGlobalStorage::WriteAll<1>(out_inverse_waveform).device_ptr(),
            x );
}


__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, DataStorageSize numElem )
{
    const int
            x = blockIdx.x*blockDim.x + threadIdx.x;

    inverse_elem( x, in_wavelet, out_inverse_waveform, numElem );
}


/*
void wtInverseEllipse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, int n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse_ellipse<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, area, n_valid_samples );
}

__global__ void kernel_inverse_ellipse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, int n_valid_samples )
{
    const int
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // disc selection
    for (int fi=0; fi<numElem.height; fi++)
    {
        float rx = area.z-area.x;
        float ry = area.w-area.y;
        float dx = x-area.x;
        float dy = fi-area.y;

        if (dx*dx/rx/rx + dy*dy/ry/ry < 1) {
            // select only the real component of the complex transform
            a += in_wavelet[ x + fi*numElem.width ].x;
        }
    }

    out_inverse_waveform[x] = a;
}

void wtInverseBox( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, int n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse_box<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, area, n_valid_samples );
}

__global__ void kernel_inverse_box( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, int n_valid_samples )
{
    const int
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // box selection
    if (x>=area.x && x<=area.z)
      {
        for (int fi=max(0.f,area.y); fi<numElem.height && fi<area.w; fi++)
        {
            float2 v = in_wavelet[ x + fi*numElem.width ];
            // select only the real component of the complex transform
            a += v.x;
        }
    }

    out_inverse_waveform[x] = a;
}
*/
void wtClamp( Tfr::ChunkData::Ptr in_wtp, size_t sample_offset, Tfr::ChunkData::Ptr out_clamped_wtp )
{
    cudaPitchedPtrType<float2> in_wt(CudaGlobalStorage::ReadOnly<2>( in_wtp ).getCudaPitchedPtr());
    cudaPitchedPtrType<float2> out_clamped_wt(CudaGlobalStorage::WriteAll<2>( out_clamped_wtp ).getCudaPitchedPtr());

    dim3 grid, block;
    int block_size = 256;
    out_clamped_wt.wrapCudaGrid2D( block_size, grid, block );

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_clamp<<<grid, block, 0>>>( in_wt, sample_offset, out_clamped_wt );
}

__global__ void kernel_clamp( cudaPitchedPtrType<float2> in_wt, size_t sample_offset, cudaPitchedPtrType<float2> out_clamped_wt )
{
    elemSize3_t writePos;
    if( !out_clamped_wt.unwrapCudaGrid( writePos ))
        return;

    elemSize3_t readPos = writePos;
    readPos.x += sample_offset;

    out_clamped_wt.e( writePos ) = in_wt.elem(readPos);
}


#endif // WAVELETKERNEL_CUDA_CU
